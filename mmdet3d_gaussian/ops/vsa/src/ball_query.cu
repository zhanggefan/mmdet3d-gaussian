#include "hip/hip_runtime.h"
/*
Stacked-batch-data version of ball query, modified from the original
implementation of official PointNet++ codes. Written by Shaoshuai Shi All Rights
Reserved 2019-2020.
*/
#include "cuda_utils.h"
#include <THC/THC.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <torch/serialize/tensor.h>

__global__ void ball_query_kernel(int B, int M, float radius, int nsample,
                                  const float *new_xyz,
                                  const int *new_xyz_batch_cnt,
                                  const float *xyz, const int *xyz_batch_cnt,
                                  int *idx) {
  // :param xyz: (N1 + N2 ..., 3) xyz coordinates of the features
  // :param xyz_batch_cnt: (batch_size), [N1, N2, ...]
  // :param new_xyz: (M1 + M2 ..., 3) centers of the ball query
  // :param new_xyz_batch_cnt: (batch_size), [M1, M2, ...]
  // output:
  //      idx: (M, nsample)
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (pt_idx >= M)
    return;

  int bs_idx = 0, pt_cnt = new_xyz_batch_cnt[0];
  for (int k = 1; k < B; k++) {
    if (pt_idx < pt_cnt)
      break;
    pt_cnt += new_xyz_batch_cnt[k];
    bs_idx = k;
  }

  int xyz_batch_start_idx = 0;
  for (int k = 0; k < bs_idx; k++)
    xyz_batch_start_idx += xyz_batch_cnt[k];
  // for (int k = 0; k < bs_idx; k++) new_xyz_batch_start_idx +=
  // new_xyz_batch_cnt[k];

  new_xyz += pt_idx * 3;
  xyz += xyz_batch_start_idx * 3;
  idx += pt_idx * nsample;

  float radius2 = radius * radius;
  float new_x = new_xyz[0];
  float new_y = new_xyz[1];
  float new_z = new_xyz[2];
  int n = xyz_batch_cnt[bs_idx];

  int cnt = 0;
  for (int k = 0; k < n; ++k) {
    float x = xyz[k * 3 + 0];
    float y = xyz[k * 3 + 1];
    float z = xyz[k * 3 + 2];
    float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
               (new_z - z) * (new_z - z);
    if (d2 < radius2) {
      if (cnt == 0) {
        for (int l = 0; l < nsample; ++l) {
          idx[l] = k;
        }
      }
      idx[cnt] = k;
      ++cnt;
      if (cnt >= nsample)
        break;
    }
  }
  if (cnt == 0)
    idx[0] = -1;
}

int ball_query(float radius, int nsample, const at::Tensor &new_xyz_tensor,
               const at::Tensor &new_xyz_batch_cnt_tensor,
               const at::Tensor &xyz_tensor,
               const at::Tensor &xyz_batch_cnt_tensor, at::Tensor &idx_tensor) {
  CHECK_INPUT(new_xyz_tensor);
  CHECK_INPUT(xyz_tensor);
  CHECK_INPUT(new_xyz_batch_cnt_tensor);
  CHECK_INPUT(xyz_batch_cnt_tensor);
  const float *new_xyz = new_xyz_tensor.data_ptr<float>();
  const float *xyz = xyz_tensor.data_ptr<float>();
  const int *new_xyz_batch_cnt = new_xyz_batch_cnt_tensor.data<int>();
  const int *xyz_batch_cnt = xyz_batch_cnt_tensor.data<int>();
  int *idx = idx_tensor.data<int>();

  hipError_t err;

  int B = xyz_batch_cnt_tensor.size(0);
  int M = new_xyz_tensor.size(0);

  dim3 blocks(DIVUP(M, THREADS_PER_BLOCK)); // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  ball_query_kernel<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
      B, M, radius, nsample, new_xyz, new_xyz_batch_cnt, xyz, xyz_batch_cnt,
      idx);
  // hipDeviceSynchronize();  // for using printf in kernel function
  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
  return 1;
}
